#include "hip/hip_runtime.h"
#include <util.h>
#include <vector>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define MAX_KMER 14
#define MIN_KMER 2





#define READ_LEN_MAX 200
#define REF_LEN_MAX 1024*1024
#define READ_CNT_MAX 1024*20


int num_kmer_in_read(char *read, int k);


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__device__ int d_strlen(const char* string){
    int length = 0;
    while (*string++)
        length++;
    return (length);
}

//Compares string until nth character, m times
//Modify with m to enable load balancing between threads
__device__ int d_strncmp( const char * s1, const char * s2, size_t n, int m )
{
    while ( n && *s1 && ( *s1 == *s2 ) && --m )
    {
        ++s1;
        ++s2;
        --n;
    }
    if ( (n == 0) && (m == 0) )
    {
        return 0;
    }
    else
    {
        return ( *(unsigned char *)s1 - *(unsigned char *)s2 );
    }
}


__global__ void match_k_mer_g(int res_arr[], int res_cnt[], int k_mer, char* read_str, char* reference_str , int read_len, int ref_len, int read_cnt)
{

	//Gets each k-mer, assigns it to a thread
	int offset2 = blockIdx.x + read_len*blockIdx.y ;
	int result;

	//Go over the ref_len, find the matching sequences
	for(int i = 0; i < ref_len; i++)
	{
		//Modified version of d_strncmp is used. It checks the first k_mer elements of the two strings
		result = d_strncmp((reference_str+i),(read_str+offset2) , k_mer, k_mer);
		
		//If there is a match, store the index i and increase the count
		if (result == 0) 
		{

			//if the index is empty or there exists a smaller index:
			if ((res_arr[offset2] == -1)||( i < res_arr[offset2] ))
			{
				//write the first matched index or smallest index, for every other match increase the count (else)
				res_arr[offset2] = i;
				res_cnt[offset2]++;
			}
			else
			{
				//printf("HERE");
				res_cnt[offset2]++;
			}
	}
	
	}

}


__host__ void match_k_mer_h(int res_arr[], int cnt[], char* ref_str, char* read_str, int k_mer, int read_len, int ref_len, int read_cnt)
{
	int* dev_result_idx_arr ;
	int* dev_res_cnt_arr ;
	char* dev_ref_str;
	char* dev_read_str;
	
	int size_ref = REF_LEN_MAX*sizeof(char);
	int size_rds = READ_LEN_MAX*READ_CNT_MAX*sizeof(char);
	int size_res = READ_CNT_MAX*(READ_LEN_MAX-MIN_KMER+1)*sizeof(int);
	int size_res_cnt = size_res;
	


	HANDLE_ERROR( hipMalloc((void**)&dev_ref_str,size_ref) );
	HANDLE_ERROR( hipMalloc((void**)&dev_read_str,size_rds) );
	HANDLE_ERROR( hipMalloc((void**)&dev_result_idx_arr,size_res) );
	HANDLE_ERROR( hipMalloc((void**)&dev_res_cnt_arr,size_res_cnt) );
	
	//Copy the result arrays, host set them to -1 (result_arr) and 0 (cnt_arr)
	HANDLE_ERROR( hipMemcpy( dev_ref_str, ref_str, size_ref, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy( dev_read_str, read_str, size_rds, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy( dev_result_idx_arr, res_arr, size_res, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy( dev_res_cnt_arr, cnt,size_res_cnt, hipMemcpyHostToDevice) );
	
	
	dim3 dimBlock(read_len-k_mer+1,read_cnt,1);
	dim3 dimThread(1,1);
	

	
	match_k_mer_g<<<dimBlock,dimThread>>> (dev_result_idx_arr, dev_res_cnt_arr, k_mer, dev_read_str, dev_ref_str, read_len, ref_len, read_cnt);
	
	
	
	HANDLE_ERROR( hipMemcpy(res_arr, dev_result_idx_arr, size_res, hipMemcpyDeviceToHost) );
	HANDLE_ERROR( hipMemcpy(cnt, dev_res_cnt_arr, size_res_cnt, hipMemcpyDeviceToHost) );
	
	

	HANDLE_ERROR( hipFree( dev_result_idx_arr ) );
	HANDLE_ERROR( hipFree( dev_res_cnt_arr ) );
    HANDLE_ERROR( hipFree( dev_ref_str ) );
    HANDLE_ERROR( hipFree( dev_read_str ) );
}





int main(int argc, char** argv)
{
    if(argc != 5) {
        printf("Wrong argments usage: ./kmer [REFERENCE_FILE] [READ_FILE] [k] [OUTPUT_FILE]\n" );
    }

    FILE *fp;
    int k;

    //malloc instead of allocating in stack
    char *reference_str = (char*) malloc(MAX_REF_LENGTH * sizeof(char));
    char *read_str = (char*) malloc(MAX_READ_LENGTH * sizeof(char));

    char *reference_filename, *read_filename, *output_filename;
    int reference_length;

    reference_filename = argv[1];
    read_filename = argv[2];
    k = atoi(argv[3]);
    output_filename = argv[4];

    fp = fopen(reference_filename, "r");
    if (fp == NULL) {
        printf("Could not open file %s!\n",reference_filename);
        return 1;
    }

    if (fgets(reference_str, MAX_REF_LENGTH, fp) == NULL) { //A single line only
        printf("Problem in file format!\n");
        return 1;
    }
    reference_str[strcspn(reference_str, "\n")] = 0; //Remove the trailing \n character
    
    reference_length = strlen(reference_str);
	
    //printf("Reference str is = %s\n", reference_str);
    fclose(fp);

    //Read queries
    StringList queries;

    initStringList(&queries, 3);  // initially 3 elements


    int success = read_file(read_filename,&queries);
    /*Comment the print
    if(success){
        for(int i = 0; i < queries.used; i++) {
            //printf("read : %s\n", queries.array[i]);
        }
    }
    */
    
   
    
    ////////////////////////////////////////////////////////////////////////
    ////////////// THIS IS A GOOD PLACE TO DO YOUR COMPUTATIONS ////////////
    ////////////////////////////////////////////////////////////////////////
    


	//Flatten the queries
	const int read_length = strlen(queries.array[0]);
	char* flat_read_str = (char*) malloc( read_length*(queries.used)*sizeof(char));
	
    for(int i = 0; i < queries.used; i++) 
    {
    	for(int j = 0; j < read_length; j++)
    	{
    		flat_read_str[i*read_length+j] = queries.array[i][j];
    	}
    }



	int* result_idx_arr ;
	int* cnt ;
	
	int len_res_arr = READ_CNT_MAX*(READ_LEN_MAX-MIN_KMER+1);
	result_idx_arr = (int*)malloc(len_res_arr*sizeof(int));
	cnt = (int*)malloc(len_res_arr*sizeof(int));
	
	//Set the result array to -1 and cnt array to 0
 	for(int i=0; i < len_res_arr ; i++)
 	{
 		result_idx_arr[i] = -1;
 		cnt[i] = 0;
 	}
	
	
	//Call Host Function
	match_k_mer_h(result_idx_arr, cnt ,reference_str, flat_read_str, k, read_length, reference_length, queries.used);

	
	
	int k_mer_cnt_arr[queries.used];
	//Reduce the count values for read sequences
    for(int i=0; i < queries.used ; i++)
 	{
 		k_mer_cnt_arr[i] = 0;
 		for(int j = 0; j < read_length-k+1; j++)
 		{
 			k_mer_cnt_arr[i] = k_mer_cnt_arr[i] +cnt[i*(read_length)+j];
 		}
 	}    
 	
 	
 	//Print the output to the file
 	FILE *fp_out;
    fp_out = fopen(output_filename, "w");
    for(int i=0; i < queries.used ; i++)
 	{
 		fprintf(fp_out,"%d",k_mer_cnt_arr[i]) ;
 		for(int j = 0; j < read_length-k+1; j++)
 		{
 			fprintf(fp_out," %d",result_idx_arr[i*(read_length)+j]) ;
 		}
 		fprintf(fp_out,"\n") ;
 		//printf("%d\n",result_idx_arr[i]) ;
 	} 
 	
 	fclose(fp_out);

    
    ////////////////////////////////////////////////////////////////////////
    ////////////// THIS IS A GOOD PLACE TO DO YOUR COMPUTATIONS ////////////
    ////////////////////////////////////////////////////////////////////////
    
    
    //Free up
    
    freeStringList(&queries);
    free(reference_str);
    free(read_str);
    free(result_idx_arr);
    
    return 0;
}


